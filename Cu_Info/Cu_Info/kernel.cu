
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main()
{
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	for (int i = 0; i < count; ++i)
	{
		hipGetDeviceProperties(&prop, i);
		cout << "---Deneral Information for device---" << i << endl;
		cout << "Name:" << prop.name << endl;
		cout << "Computer capablility:" << prop.major << prop.minor << endl;
		cout << "Clock rate:" << prop.clockRate << endl;
		cout << "Device copy overlap:";
		if (prop.deviceOverlap)
			cout << "Enabled" << endl;
		else
			cout << "Disabled" << endl;
		cout << endl;
		cout << "---Memory information for device---" << i << endl;
		cout << "Total global mem:" << prop.totalGlobalMem << endl;
		cout << "Total constant mem:" << prop.totalConstMem << endl;
		cout << "Max mem pitch:" << prop.memPitch << endl;
		cout << "Texture alignment:" << prop.textureAlignment << endl;
		cout << endl;
		cout << "---MP informations for device---" << i << endl;
		cout << "Shared mem per mp:" << prop.sharedMemPerBlock << endl;
		cout << "Registers per mp:" << prop.regsPerBlock << endl;
		cout << "Threads in warp:" << prop.warpSize << endl;
		cout << "Max threads per block:" << prop.maxThreadsPerBlock << endl;
		cout << "Max thread dimensions:" << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << endl;
		cout << "Max grid dimensions:" << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;
	}
	system("pause");
	return 0;
}