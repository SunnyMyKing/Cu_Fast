#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <Windows.h>

#define LINE 2048  //2048*2048��ʱ��0.013704s     4098*4096��ʱ��0.05545s
#define BLOCK_SIZE 16

__global__ void matrix_kernel_1(float* dev_c, const float* dev_a, const float *dev_b)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	
	int aStart = LINE*(by*BLOCK_SIZE);   
	int aEnd = aStart + LINE - 1;
	int aDiff = BLOCK_SIZE;

	int bStart = BLOCK_SIZE*bx;
	int bDiff = BLOCK_SIZE*LINE;

	float cSub = 0;
	for (int a = aStart, b = bStart; a <= aEnd; a += aDiff, b += bDiff)
	{
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[ty][tx] = dev_a[a + LINE*ty + tx];
		Bs[tx][ty] = dev_b[b + LINE*ty + tx];

		//��B�������ת��
		/*float TempSwap;
		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			for (int j = 0; j < BLOCK_SIZE; j++)
			{
				TempSwap = Bs[i][j];
				Bs[i][j] = Bs[j][i];
				Bs[j][i] = TempSwap;
			}
		}*/

		__syncthreads();

		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			cSub += As[ty][k] * Bs[tx][k];
		}

		__syncthreads();
	}

	int Index = (by*BLOCK_SIZE + ty)*LINE + (bx*BLOCK_SIZE + tx);
	dev_c[Index] = cSub;
}

int main()
{
	LARGE_INTEGER tc, start, stop;
	float *Matrix_A;
	float *Matrix_B;
	float *Matrix_C;
	Matrix_A = (float *)malloc(sizeof(float) * LINE * LINE);
	Matrix_B = (float *)malloc(sizeof(float) * LINE * LINE);
	Matrix_C = (float *)malloc(sizeof(float) * LINE * LINE);
	for (int i = 0; i < LINE * LINE; i++)
	{
		//Matrix_A[i] = std::rand() % 1000;
		//Matrix_B[i] = std::rand() % 1000;
		Matrix_A[i] = i % 4;
		Matrix_B[i] = i % 4;
	}
	float *dev_m_A;
	float *dev_m_B;
	float *dev_m_C;

	//std::chrono::system_clock::time_point GPU_start = std::chrono::system_clock::now();
	hipMalloc((void **)(&dev_m_A), LINE *LINE*sizeof(float));
	
	hipMalloc((void **)(&dev_m_B), LINE * LINE *sizeof(float));
	
	hipMalloc((void **)(&dev_m_C), LINE*LINE*sizeof(float));
	
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&start);
	hipMemcpy(dev_m_A, Matrix_A, sizeof(float) * LINE * LINE, hipMemcpyHostToDevice);
	hipMemcpy(dev_m_B, Matrix_B, sizeof(float) * LINE * LINE, hipMemcpyHostToDevice);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(LINE / BLOCK_SIZE, LINE / BLOCK_SIZE);
	//Calculate_Matrix <<<blocks, threads >>>(dev_m_A, dev_m_B, dev_m_C);
	matrix_kernel_1 <<<blocks, threads >>>(dev_m_C, dev_m_A, dev_m_B);
	hipMemcpy(Matrix_C, dev_m_C, sizeof(float) * LINE *LINE, hipMemcpyDeviceToHost);
	QueryPerformanceCounter(&stop);
	
	printf("Use Time:%f\n", (stop.QuadPart - start.QuadPart)*1.0 / tc.QuadPart);
	//std::chrono::system_clock::time_point GPU_end = std::chrono::system_clock::now();
	//std::cout << double(std::chrono::duration_cast<std::chrono::microseconds>(GPU_end - GPU_start).count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << "s" << std::endl;
	/*for (int i = 0; i < LINE; i++)
	{
		std::cout << Matrix_C[i*LINE + i] << " ";
	}
	std::cout << std::endl;*/
	hipFree(dev_m_A);
	hipFree(dev_m_B);
	hipFree(dev_m_C);
	free(Matrix_A);
	free(Matrix_B);
	free(Matrix_C);

	system("pause");
	return 0;
}

